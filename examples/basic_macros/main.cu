
#include <hip/hip_runtime.h>
#include <cstdio>

#define CUDA_CHECK(expr)                                                \
  do {                                                                  \
    hipError_t err = (expr);                                           \
    if (err != hipSuccess) {                                           \
      fprintf(stderr, "CUDA Error Code  : %d\n     Error String: %s\n", \
              err, hipGetErrorString(err));                            \
      exit(err);                                                        \
    }                                                                   \
  } while (0)

__global__ void kernel() {
  printf("cuda kernel called!\n");
}

void launch() {
  kernel<<<1, 1>>>();
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());
}

int main() {
  launch();
  return 0;
}
